#include "hip/hip_runtime.h"
typedef struct {
  float value;
  int64_t index;
} pair;

extern "C"
__global__ void topkspspmm(
  const int64_t* __restrict__ pACrowInds, // [m + 1]
  const int64_t* __restrict__ pAColInds,  // [nnz_a]
  const float* __restrict__ pAVals,       // [nnz_b]

  const int64_t* __restrict__ pBCrowInds, // [n + 1]
  const int64_t* __restrict__ pBColInds,  // [nnz_b]
  const float* __restrict__ pBVals,       // [nnz_b]

  const int64_t* __restrict__ pAlpha1,   // [2]
  const int64_t* __restrict__ pAlpha2,   // [2]
  const int64_t* __restrict__ pBeta1,    // [2]
  const int64_t* __restrict__ pBeta2,    // [2]
  const int64_t* __restrict__ pPrime1,   // [2]
  const int64_t* __restrict__ pPrime2,   // [2]

  int64_t* pTopkInds, //[m, nCands]
  float* pTopkVals, //[m, nCands]

  int m, int n, int k
) {
  constexpr int TPB = _TPB_;   // threads per block
  constexpr int MaxNNZPR = _MAXNNZPR_; // max number of nonzero elements per row
  constexpr int StackCap = _STACKCAP_; // stack capacity used for sorting
  constexpr int TileM = _TILEM_; // number of rows from A matrix
  constexpr int TileN = TPB; // number of rows from B matrix to load at each iteration
  
  constexpr int ThreadsPerGroup = 32; //number of threads per thread group
  constexpr int NumGroups = TPB / ThreadsPerGroup; //number of thread groups
  constexpr int GroupTileN = TileN / NumGroups; //number of rows from B matrix per group
  constexpr int NumBuckets = _NUMBUCKETS_; // number of hashmap buckets
  // constexpr int NumBuckets = TileM * MaxNNZPR; // number of hashmap buckets
  
  int tid = threadIdx.x;
  int mStart = blockIdx.x * TileM;
  int gx = tid % ThreadsPerGroup;
  int gy = tid / ThreadsPerGroup;

  extern __shared__ int64_t smemPtr[];
  // SmemTensor2D<int64_t, TileM, MaxNNZPR> smemAColInds(smemPtr);
  // SmemTensor2D<float, TileM, MaxNNZPR> smemAVals(smemPtr);
  SmemTensor2D<int64_t, NumBuckets, 1> smemHashmapKeys(smemPtr);  // [NumBuckets, 1]
  SmemTensor2D<float, NumBuckets, 1> smemHashmapVals(smemHashmapKeys.endPtr); //[NumBuckets, 1]

  SmemTensor1D<int64_t, TPB> smemIndexExchange(smemHashmapVals.endPtr);  //[TPB]
  SmemTensor1D<float, TPB> smemValueExchange(smemIndexExchange.endPtr);  //[TPB]
  
  SmemTensor1D<int, TileM> smemSortTrigger(smemValueExchange.endPtr); //[TileM]
  SmemTensor1D<float, TileM> smemMinValueExchange(smemSortTrigger.endPtr); //[TileM]

  #pragma unroll
  for (int i=0; i<TileM; i++){
    smemSortTrigger.set(i, 0);
  }

  // initialize hashmap
  SmemHashmap<int64_t, float, 1, 1, NumBuckets, TPB> hashmap(
    pAlpha1, pAlpha2,
    pBeta1,  pBeta2, 
    pPrime1, pPrime2,
    smemHashmapKeys,
    smemHashmapVals,
    -1
  );

  // initialize stack
  Stack<pair, StackCap> threadTopkStack[TileM];
  #pragma unroll
  for (int i=0; i<TileM; i++){
    pair empty_pair = { -INFINITY, -1 };
    threadTopkStack[i].fill(empty_pair);
  }
  float threadMinValue[TileM] = { -INFINITY };
  float topkVal[TileM] = { -INFINITY };
  int64_t topkInd[TileM] = { -2 };

  // load tile from A
  #pragma unroll
  for (int i=0; i<TileM; i++){
    int iM = mStart + i;
    if (iM < m){
      int aRowStart = pACrowInds[iM];
      int aRowEnd = pACrowInds[iM + 1];
      int aRowNNZ = aRowEnd - aRowStart;
      #pragma unroll
      for (int j=0; j < div_ru(MaxNNZPR, TPB); j++){
        #pragma unroll
        for (int t=0; t<TPB; t++){
          if (tid == 0){
            int iNZ = j * TPB + t;
            if (iNZ < MaxNNZPR && iNZ < aRowNNZ){
              int64_t iK = pAColInds[aRowStart + iNZ];
              float value[1] = { pAVals[aRowStart + iNZ] };
              int64_t key[1] = { iM * k + iK };
              hashmap.set(key, value);
            }
          }
        }
        // int iNZ = j * TPB + tid;
        // if (iNZ < MaxNNZPR && iNZ < aRowNNZ){
        //   int64_t iK = pAColInds[aRowStart + iNZ];
        //   float value[1] = { pAVals[aRowStart + iNZ] };
        //   int64_t key[1] = { iM * k + iK };
        //   hashmap.set(key, value);
        // }
      }
    }
  }
  __syncthreads();
  
  for (int a = 0; a < div_ru(n, TPB); a++){
    int64_t iN = a * TPB + tid;
    // pTopkInds[(mStart) * TPB + tid] = iN;
    int64_t bRowStart = 0;
    int64_t bRowNNZ = 0;
    float accumulator[TileM] = {0.f};
    if (iN < n){
      bRowStart = pBCrowInds[iN];
      bRowNNZ = pBCrowInds[iN + 1];
    }
    for (int b = 0; b < ThreadsPerGroup; b++){
      int64_t iN2 = a * TPB + gy * ThreadsPerGroup + b;
      int64_t curBRowStart = __shfl_sync(0xffffffff, bRowStart, b);
      int64_t curBRowNNZ = __shfl_sync(0xffffffff, bRowNNZ, b);

      for (int c = 0; c < div_ru(curBRowNNZ, ThreadsPerGroup); c++){
        int iNZ = c * ThreadsPerGroup + gx;
        int64_t iK = -1;
        float bVal = 0.f;
        if (iNZ < curBRowNNZ){
          iK = pBColInds[curBRowStart + iNZ];
          bVal = pBVals[curBRowStart + iNZ];
        }
        #pragma unroll
        for (int d = 0; d < TileM; d++){
          int iM = mStart + d;
          int64_t key[1] = { iM * k + iK };
          float value[1] = { 0.f };
          if (iK != -1){
            bool isFound = hashmap.get(key, value);
          }
          float aVal = value[0];
          float cVal = aVal * bVal;
          warp_sum<float, 32>(cVal);
          // cVal = __shfl_sync(0xffffffff, cVal, 0);
          if (gx == b){
            accumulator[d] += cVal;
          }
        }
      }
    }
    #pragma unroll
    for (int i=0; i<TileM; i++){
      int iM = mStart + i;
      if (iM >= m) continue;
      int candIndex = tid;
      pTopkVals[iM * TPB + candIndex] = accumulator[i];
      pTopkInds[iM * TPB + candIndex] = iN;
    }
    // pTopkInds[(mStart) * TPB + tid] = iN;
    // push index value pair into stack
    __syncthreads();
    pair oldPairs[TileM];
    #pragma unroll
    for (int i = 0; i < TileM; i++){
      oldPairs[i] = { -INFINITY, -3 };
      if (threadTopkStack[i].is_full()){
        threadTopkStack[i].pop(oldPairs[i]);
        if (oldPairs[i].value > threadMinValue[i]){
          smemSortTrigger.set(i, 1);
        }
      }

      pair newPair;
      if (iN < n){
        newPair = { accumulator[i],  iN};
      } else {
        newPair = { -INFINITY, -4 };
      }
      
      if (accumulator[i] > threadMinValue[i]){
        threadTopkStack[i].push(newPair);
      }
    }
    __syncthreads();

    // sort if necessary
    #pragma unroll
    for (int i=0; i<TileM; i++){
      if (smemSortTrigger.get(i) > 0){
        __syncthreads();
        bitonic_sort<TPB>(oldPairs[i].value, oldPairs[i].index, 
                          smemValueExchange.startPtr, smemIndexExchange.startPtr, 
                          tid);

        bitonic_sort_global<TPB>(topkVal[i], topkInd[i], 
                                oldPairs[i].value, oldPairs[i].index, 
                                smemValueExchange.startPtr, smemIndexExchange.startPtr,
                                tid);
        __syncthreads();
        if (tid == TPB - 1){
          smemMinValueExchange.set(i, topkVal[i]);
        }
        __syncthreads();
        threadMinValue[i] = smemMinValueExchange.get(i);
      }
    }
    __syncthreads();
  }
  // sort the remaining items in stack
  #pragma unroll
  for (int i=0; i<TileM; i++){
    smemSortTrigger.set(i, 0);
    __syncthreads();

    #pragma unroll
    for (int j=0; j<StackCap; j++){
      pair oldPair = { -INFINITY, -5 };
      if (!threadTopkStack[i].is_empty()){
         threadTopkStack[i].pop(oldPair);
         if (oldPair.value > threadMinValue[i]){
           smemSortTrigger.set(i, 1);
         }
      }
      __syncthreads();

      if (smemSortTrigger.get(i) > 0){
        __syncthreads();
        bitonic_sort<TPB>(oldPair.value, oldPair.index, 
                          smemValueExchange.startPtr, smemIndexExchange.startPtr, 
                          tid);

        bitonic_sort_global<TPB>(topkVal[i], topkInd[i], 
                                oldPair.value, oldPair.index, 
                                smemValueExchange.startPtr, smemIndexExchange.startPtr,
                                tid);
        __syncthreads();
        smemSortTrigger.set(i, 0);
        if (tid == TPB - 1){
          smemMinValueExchange.set(i, topkVal[i]);
        }
        __syncthreads();
        threadMinValue[i] = smemMinValueExchange.get(i);
      }
      __syncthreads();
    }
  }

  // write results back
  #pragma unroll
  for (int i=0; i<TileM; i++){
    int iM = mStart + i;
    if (iM >= m) continue;
    int candIndex = tid;
    // pTopkVals[iM * TPB + candIndex] = topkVal[i];
    // pTopkInds[iM * TPB + candIndex] = topkInd[i];
  }
}