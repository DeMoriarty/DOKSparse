#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"
#include <stdio.h>
// #include "mma"

#define likely(x)      __builtin_expect(!!(x), 1)
#define unlikely(x)    __builtin_expect(!!(x), 0)
#define load(x)        __ldcg(x)
#define store(x, value) __stcs(x, value)
#define div_ru(a, b) (a + b - 1) / b 
#define div_rd(a, b) a / b 
#define VOLATILE
#ifndef INFINITY
#define INFINITY __int_as_float(0x7f800000)
#endif
#define DEBUG

#ifdef DEBUG
#define COMPILER_ASSERT(EXPRESSION)   switch (0) {case 0: case (EXPRESSION):;}
#else
#define COMPILER_ASSERT(EXPRESSION)
#endif

#define CUDA_DEVICE_INLINE __device__ __forceinline__

// #define LAYOUT_C true
// #define LAYOUT_F false
// #define TRANSFORM_N true
// #define TRANSFORM_T false

// typedef bool MATRIX_LAYOUT;
// typedef bool MATRIX_TRANSFORM;
typedef unsigned char uint8_t;
typedef long long ll_t;
typedef unsigned long long ull_t;

typedef struct __builtin_align__(8){
  half x1, x2, x3, x4;
} half4;

typedef struct __builtin_align__(16){
  half x1, x2, x3, x4, x5, x6, x7, x8;
} half8;

typedef struct __builtin_align__(16){
  unsigned char x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15, x16;
} uchar16;

typedef struct __builtin_align__(16){
  char x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15, x16;
} char16;

typedef struct {
  int x;
} Coord1D;

typedef struct {
  int x, y;
} Coord2D;

typedef struct {
  int x, y, z;
} Coord3D;

typedef struct {
  int x, y, z, t;
} Coord4D;

typedef struct {
  int x, y, z, t, u;
} Coord5D;

typedef union {
  int as_int32[1];
  unsigned int as_uint32[1];
  short as_int16[2];
  unsigned short as_uint16[2];
  signed char as_int8[4];
  unsigned char as_uint8[4]; 
  float as_float[1];
  half2 as_half2[1];
  half as_half[2];  
} Data4B;

typedef union {
  long long as_int64[1];
  unsigned long long as_uint64[1];
  int as_int32[2];
  unsigned int as_uint32[2];
  short as_int16[4];
  unsigned short as_uint16[4];
  signed char as_int8[8];
  unsigned char as_uint8[8]; 
  double as_double[1];
  half4 as_half4[1];
  float2 as_float2[1];
  float as_float[2];
  half2 as_half2[2];
  half as_half[4];  
} Data8B;

typedef union {
  uchar16 as_uchar16[1];
  char16 as_char16[1];
  long long as_int64[2];
  unsigned long long as_uint64[2];
  int as_int32[4];
  unsigned int as_uint32[4];
  short as_int16[8];
  unsigned short as_uint16[8];
  signed char as_int8[16];
  unsigned char as_uint8[16];
  half8 as_half8[1]; 
  double as_double[2];
  half4 as_half4[2];
  float2 as_float2[2];
  float as_float[4];
  half2 as_half2[4];
  half as_half[8];  
} Data16B;


template <typename MutexType>
CUDA_DEVICE_INLINE
void mutex_lock_thread(
  MutexType *mutex,
  const MutexType onValue,
  const MutexType offValue
) {
  unsigned int ns = 8;
  unsigned int counter = 0;
  while (atomicCAS(mutex, offValue, onValue) == onValue) {
    __nanosleep(ns);
    counter ++;
    if (counter > 1000) break;
    if (ns < 256) {
      ns *= 2;
    }
  }
}

template <typename MutexType>
CUDA_DEVICE_INLINE
void mutex_unlock_thread(
  MutexType *mutex,
  const MutexType offValue
) {
  __threadfence();
  atomicExch(mutex, offValue);
  __threadfence();
}

CUDA_DEVICE_INLINE
long long atomicCAS(
  ll_t *address,
  ll_t compare,
  ll_t val
){
  ull_t old = atomicCAS(
    reinterpret_cast<ull_t*>(address),
    reinterpret_cast<ull_t&>(compare),
    reinterpret_cast<ull_t&>(val)
  );
  return reinterpret_cast<ll_t&>(old);
}
template <
  typename T
>
class SmemTensor0D{
  public:
    VOLATILE T* endPtr;
    VOLATILE T* startPtr;

    CUDA_DEVICE_INLINE
    SmemTensor0D(VOLATILE void* smemPtr)  
        : startPtr(reinterpret_cast<T*>(smemPtr))
        , endPtr(reinterpret_cast<T*>(smemPtr))
    {
    }

    CUDA_DEVICE_INLINE
    T get(){
      return startPtr[0];
    }

    CUDA_DEVICE_INLINE
    T* get_ptr(){
      return startPtr;
    }

    CUDA_DEVICE_INLINE
    void set(T value){
      startPtr[0] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U get_reinterpreted(){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return newPtr[0];
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    void set_reinterpreted(U value){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      newPtr[0] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U* get_ptr_reinterpreted(){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return &newPtr[0];
    }
};

template <
  typename T,
  int ShapeX
>
class SmemTensor1D{
  public:
    VOLATILE T* endPtr;
    VOLATILE T* startPtr;

    CUDA_DEVICE_INLINE
    SmemTensor1D(VOLATILE void* smemPtr)  
        : startPtr(reinterpret_cast<T*>(smemPtr))
        , endPtr(reinterpret_cast<T*>(smemPtr) + shape().x)
    {
    }

    CUDA_DEVICE_INLINE
    T get(int x){
      return startPtr[x];
    }

    CUDA_DEVICE_INLINE
    T* get_ptr(int x){
      return &startPtr[x];
    }

    CUDA_DEVICE_INLINE
    void set(int x, T value){
      startPtr[x] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U get_reinterpreted(int x){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return newPtr[x];
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    void set_reinterpreted(int x, U value){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      newPtr[x] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U* get_ptr_reinterpreted(int x){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return &newPtr[x];
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord1D shape(){
      return { ShapeX };
    }
};

template <
  typename T,
  int ShapeX,
  int ShapeY
>
class SmemTensor2D{
  public:
    VOLATILE T* endPtr;
    VOLATILE T* startPtr;

    CUDA_DEVICE_INLINE
    SmemTensor2D(VOLATILE void* smemPtr)  
        : startPtr(reinterpret_cast<T*>(smemPtr))
        , endPtr(reinterpret_cast<T*>(smemPtr) + shape().x * shape().y)
    {
    }

    CUDA_DEVICE_INLINE
    T get(int x, int y){
      return startPtr[x * stride().x + y];
    }

    CUDA_DEVICE_INLINE
    T* get_ptr(int x, int y){
      return &startPtr[x * stride().x + y];
    }

    CUDA_DEVICE_INLINE
    void set(int x, int y, T value){
      startPtr[x * stride().x + y] = value;
    }

    CUDA_DEVICE_INLINE
    SmemTensor1D<T, ShapeY> get_child(int x){
      SmemTensor1D<T, ShapeY> child(
        &startPtr[x * stride().x]
      );
      return child;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U get_reinterpreted(int x, int y){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return newPtr[
        (x * stride().x) * sizeof(T) / sizeof(U) + 
        y
      ];
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    void set_reinterpreted(int x, int y, U value){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      newPtr[
        (x * stride().x) * sizeof(T) / sizeof(U) + 
        y
      ] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U* get_ptr_reinterpreted(int x, int y){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return &newPtr[
        (x * stride().x) * sizeof(T) / sizeof(U) + 
        y
      ];
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord2D shape(){
      return {ShapeX, ShapeY};
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord1D stride(){
      return {ShapeY};
    }

};

template <
  typename T,
  int ShapeX,
  int ShapeY,
  int ShapeZ
>
class SmemTensor3D{
  public:
    VOLATILE T* endPtr;
    VOLATILE T* startPtr;

    CUDA_DEVICE_INLINE
    SmemTensor3D(VOLATILE void* smemPtr)  
        : startPtr(reinterpret_cast<T*>(smemPtr))
        , endPtr(reinterpret_cast<T*>(smemPtr) + shape().x * shape().y * shape().z)
    {
    }

    CUDA_DEVICE_INLINE
    T get(int x, int y, int z){
      return startPtr[x * stride().x + y * stride().y + z];
    }

    CUDA_DEVICE_INLINE
    T* get_ptr(int x, int y, int z){
      return &startPtr[x * stride().x + y * stride().y + z];
    }

    CUDA_DEVICE_INLINE
    void set(int x, int y, int z, T value){
      startPtr[x * stride().x + y * stride().y + z] = value;
    }

    CUDA_DEVICE_INLINE
    SmemTensor2D<T, ShapeY, ShapeZ> get_child(int x){
      SmemTensor2D<T, ShapeY, ShapeZ> child(
        &startPtr[x * stride().x]
      );
      return child;
    }

    CUDA_DEVICE_INLINE
    SmemTensor1D<T, ShapeZ> get_child(int x, int y){
      SmemTensor1D<T, ShapeZ> child(
        &startPtr[x * stride().x + y * stride().y]
      );
      return child;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U get_reinterpreted(int x, int y, int z){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return newPtr[
        (x * stride().x +  
        y * stride().y) * sizeof(T) / sizeof(U) + 
        z
      ];
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    void set_reinterpreted(int x, int y, int z, U value){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      newPtr[
        (x * stride().x +  
        y * stride().y) * sizeof(T) / sizeof(U) + 
        z
      ] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U* get_ptr_reinterpreted(int x, int y, int z){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return &newPtr[
        (x * stride().x +  
        y * stride().y) * sizeof(T) / sizeof(U) + 
        z
      ];
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord3D shape(){
      return {ShapeX, ShapeY, ShapeZ};
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord2D stride(){
      return {ShapeY * ShapeZ, ShapeZ};
    }

};

template <
  typename T,
  int ShapeX,
  int ShapeY,
  int ShapeZ,
  int ShapeT
>
class SmemTensor4D{
  public:
    VOLATILE T* endPtr;
    VOLATILE T* startPtr;
    // const Coord3D _stride;
    // const Coord4D _shape;

    CUDA_DEVICE_INLINE
    SmemTensor4D(VOLATILE void* smemPtr)  
        : startPtr(reinterpret_cast<T*>(smemPtr))
        , endPtr(&reinterpret_cast<T*>(smemPtr)[shape().x * shape().y * shape().z * shape().t])
    {
    }

    CUDA_DEVICE_INLINE
    T get(int x, int y, int z, int t){
      return startPtr[
        x * stride().x + 
        y * stride().y + 
        z * stride().z +
        t
      ];
    }

    CUDA_DEVICE_INLINE
    T* get_ptr(int x, int y, int z, int t){
      return &startPtr[
        x * stride().x + 
        y * stride().y + 
        z * stride().z +
        t
      ];
    }

    CUDA_DEVICE_INLINE
    void set(int x, int y, int z, int t, T value){
      startPtr[
        x * stride().x + 
        y * stride().y + 
        z * stride().z +
        t
      ] = value;
    }

    CUDA_DEVICE_INLINE
    SmemTensor3D<T, ShapeY, ShapeZ, ShapeT> get_child(int x){
      SmemTensor3D<T, ShapeY, ShapeZ, ShapeT> child(
        &startPtr[x * stride().x]
      );
      return child;
    }

    CUDA_DEVICE_INLINE
    SmemTensor2D<T, ShapeZ, ShapeT> get_child(int x, int y){
      SmemTensor2D<T, ShapeZ, ShapeT> child(
        &startPtr[x * stride().x + y * stride().y]
      );
      return child;
    }

    CUDA_DEVICE_INLINE
    SmemTensor1D<T, ShapeT> get_child(int x, int y, int z){
      SmemTensor1D<T, ShapeT> child(
        &startPtr[x * stride().x + y * stride().y + z * stride().z]
      );
      return child;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U get_reinterpreted(int x, int y, int z, int t){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return newPtr[
        (x * stride().x +  
        y * stride().y +  
        z * stride().z) * sizeof(T) / sizeof(U) + 
        t
      ];
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    void set_reinterpreted(int x, int y, int z, int t, U value){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      newPtr[
        (x * stride().x +  
        y * stride().y +  
        z * stride().z) * sizeof(T) / sizeof(U) + 
        t
      ] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U* get_ptr_reinterpreted(int x, int y, int z, int t){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return &newPtr[
        (x * stride().x +  
        y * stride().y +  
        z * stride().z) * sizeof(T) / sizeof(U) + 
        t
      ];
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord4D shape(){
      return {ShapeX, ShapeY, ShapeZ, ShapeT};
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord3D stride(){
      return {
        ShapeY * ShapeZ * ShapeT, 
        ShapeZ * ShapeT, 
        ShapeT
      };
    }
};
#define EMPTY 1
#define FOUND 2
#define NOT_FOUND 3
#define STORED 4
#define NOT_STORED 5

template <
  typename KeyType,
  typename ValueType,
  int KeySize,
  int ValueSize
>
class ClosedHashmap{
  private:
    ll_t _prime1[KeySize];
    ll_t _prime2[KeySize];
    ll_t _alpha1[KeySize];
    ll_t _alpha2[KeySize];
    ll_t _beta1[KeySize];
    ll_t _beta2[KeySize];
    KeyType *_pAllKeys;
    ValueType *_pAllValues;
    ll_t *_pAllUUIDs;
    ll_t _numBuckets;
    ll_t _emptyMarker;

  public:
    ll_t keyPerm[KeySize];

    CUDA_DEVICE_INLINE
    ClosedHashmap(const ll_t* pPrime1,
                  const ll_t* pPrime2,
                  const ll_t* pAlpha1,
                  const ll_t* pAlpha2,
                  const ll_t* pBeta1,
                  const ll_t* pBeta2,
                  const ll_t* pKeyPerm,
                  KeyType* pAllKeys,
                  ValueType* pAllValues,
                  ll_t* pAllUUIDs,
                  ll_t numBuckets,
                  ll_t emptyMarker
                  )
                  : _pAllKeys(pAllKeys)
                  , _pAllValues(pAllValues)
                  , _pAllUUIDs(pAllUUIDs)
                  , _numBuckets(numBuckets)
                  , _emptyMarker(emptyMarker)
    {
      #pragma unroll
      for (int i=0; i < KeySize; i++){
        keyPerm[i] = pKeyPerm[i];
        _prime1[i] = pPrime1[keyPerm[i]];
        _prime2[i] = pPrime2[keyPerm[i]];
        _alpha1[i] = pAlpha1[keyPerm[i]];
        _alpha2[i] = pAlpha2[keyPerm[i]];
        _beta1[i] = pBeta1[keyPerm[i]];
        _beta2[i] = pBeta2[keyPerm[i]];
      }
    }

    CUDA_DEVICE_INLINE
    ll_t get_hash(KeyType key[KeySize]){
      ll_t hash_code = ( (ll_t) key[0] * _alpha1[0] + _beta1[0]) % _prime1[0];
      #pragma unroll
      for (int i=1; i<KeySize; i++){
        hash_code *= ( (ll_t) key[i] * _alpha1[i] + _beta1[i]) % _prime1[i];
      }
      hash_code = llabs(hash_code);
      return hash_code;
    }

    CUDA_DEVICE_INLINE
    ll_t get_uuid(KeyType key[KeySize]){
      ll_t uuid = ( (ll_t) key[0] * _alpha2[0] + _beta2[0]) % _prime2[0];
      #pragma unroll
      for (int i=1; i<KeySize; i++){
        uuid *= ( (ll_t) key[i] * _alpha2[i] + _beta2[i]) % _prime2[i];
      }
      uuid = llabs(uuid);
      return uuid;
    }

    CUDA_DEVICE_INLINE
    bool are_keys_equal(KeyType key1[KeySize], KeyType key2[KeySize]){
      bool isEqual = key1[0] == key2[0];
      #pragma unroll
      for (int i=0; i<KeySize; i++){
        isEqual = isEqual && (key1[i] == key2[i]);
      }
      return isEqual;
    }

    CUDA_DEVICE_INLINE
    void get_key(ll_t address, KeyType key[KeySize]){
      #pragma unroll
      for (int i=0; i<KeySize; i++){
        key[i] = _pAllKeys[address * KeySize + i];
      }
    }

    CUDA_DEVICE_INLINE
    void get_key_permuted(ll_t address, KeyType key[KeySize]){
      #pragma unroll
      for (int i=0; i<KeySize; i++){
        key[i] = _pAllKeys[address * KeySize + keyPerm[i]];
      }
    }

    CUDA_DEVICE_INLINE
    void set_key(ll_t address, KeyType key[KeySize]){
      #pragma unroll
      for (int i=0; i<KeySize; i++){
        _pAllKeys[address * KeySize + i] = key[i];
      }
    
    }

    CUDA_DEVICE_INLINE
    void set_key_permuted(ll_t address, KeyType key[KeySize]){
      #pragma unroll
      for (int i=0; i<KeySize; i++){
        _pAllKeys[address * KeySize + keyPerm[i] ] = key[i];
      }
    }

    CUDA_DEVICE_INLINE
    void get_value(ll_t address, ValueType value[ValueSize]){
      #pragma unroll
      for (int i=0; i<ValueSize; i++){
        value[i] = _pAllValues[address * ValueSize + i];
      }
    }

    CUDA_DEVICE_INLINE
    void set_value(ll_t address, ValueType value[ValueSize]){
      #pragma unroll
      for (int i=0; i<ValueSize; i++){
        _pAllValues[address * ValueSize + i] = value[i];
      }
    }

    CUDA_DEVICE_INLINE
    void get_uuid(ll_t address, ll_t &uuid){
      uuid = _pAllUUIDs[address];
    }

    CUDA_DEVICE_INLINE
    ll_t get_uuid(ll_t address){
      return _pAllUUIDs[address];
    }

    CUDA_DEVICE_INLINE
    void set_uuid(ll_t address, ll_t uuid){
      _pAllUUIDs[address] = uuid;
    }

    CUDA_DEVICE_INLINE
    bool set_uuid_if_empty(int address, ll_t uuid, ll_t &oldUUID){
      ll_t *ptr = &_pAllUUIDs[address];
      // if the value at `ptr` is equal to `_emptyMarker`, then set the value of that pointer to `uuid`, return true
      // else, return false
      oldUUID = atomicCAS(ptr, _emptyMarker, uuid);
      if ( oldUUID != _emptyMarker){
        return false;
      }
      return true;
    }

    CUDA_DEVICE_INLINE
    int get_by_uuid(ll_t address, ll_t uuid, ValueType value[ValueSize]){
      ll_t candidateUUID = get_uuid(address);
      // check if the candidateKey is emptyKey
      bool isEmpty = candidateUUID == _emptyMarker;
      // is so, return not found
      if (isEmpty){
        return EMPTY;
      }
      // check if the candidateKey is equal to key
      bool isFound = candidateUUID == uuid;
      // if so, return found
      if (isFound){
        get_value(address, value);
        return FOUND;
      }
      return NOT_FOUND;
    }

     CUDA_DEVICE_INLINE
    int set_by_uuid(int address, ll_t uuid, KeyType key[KeySize], ValueType value[ValueSize]){
      // is so, store key and value in this address
      // set key to that address, if storing failed (because of another thread using that address ), return not stored
      ll_t candidateUUID;
      bool isSuccessful = set_uuid_if_empty(address, uuid, candidateUUID);
      if (isSuccessful){
        set_key(address, key);
        set_value(address, value);
        return STORED;
      }
      // check if the candidateUUID is equal to uuid
      bool isFound = uuid == candidateUUID;
      // if so, return stored
      if (isFound){
        set_key(address, key);
        set_value(address, value);
        return STORED;
      }
      // otherwise, return not found
      return NOT_STORED;
    }

    CUDA_DEVICE_INLINE
    bool get(
      KeyType key[KeySize],
      ValueType value[ValueSize],
      ValueType fallbackValue[ValueSize]
    ){
      // permute_key(key);
      ll_t hashCode = get_hash(key);
      ll_t uuid = get_uuid(key);
      #pragma unroll 2
      for (ll_t i=0; i < _numBuckets; i++){
        ll_t address = (hashCode + i) % _numBuckets;
        ll_t candidateUUID = get_uuid(address);
        // check if the candidateKey is emptyKey
        bool isEmpty = candidateUUID == _emptyMarker;
        // is so, return not found
        if (isEmpty){
          break;
        }
        // check if the candidateKey is equal to key
        bool isFound = candidateUUID == uuid;
        // if so, return found
        if (isFound){
          get_value(address, value);
          return true;
        }
      }
      #pragma unroll
      for (int j=0; j<ValueSize; j++){
        value[j] = fallbackValue[j];
      }
      return false;
    }

    CUDA_DEVICE_INLINE
    bool set(
      KeyType key[KeySize],
      ValueType value[ValueSize]
    ){
      // permute_key(key);
      ll_t hashCode = get_hash(key);
      ll_t uuid = get_uuid(key);
      #pragma unroll 2
      for (ll_t i=0; i<_numBuckets; i++){
        ll_t address = (hashCode + i) % _numBuckets;
        ll_t candidateUUID;
        bool isSuccessful = set_uuid_if_empty(address, uuid, candidateUUID);
        if (isSuccessful){
          set_key_permuted(address, key);
          set_value(address, value);
          return true;
        }
        // check if the candidateUUID is equal to uuid
        bool isFound = uuid == candidateUUID;
        // if so, return stored
        if (isFound){
          set_key_permuted(address, key);
          set_value(address, value);
          return true;
        }
      }
      return false;
    }

    template <int BatchSize>
    CUDA_DEVICE_INLINE
    void get_batched(
      KeyType key[BatchSize][KeySize],
      ValueType value[BatchSize][ValueSize],
      ValueType fallbackValue[ValueSize],
      bool isFound[BatchSize]
    ){
      ll_t hashCode[BatchSize];
      ll_t uuid[BatchSize];
      bool isDone[BatchSize];
      ll_t address[BatchSize];
      #pragma unroll
      for (int b = 0; b < BatchSize; b++){
        hashCode[b] = get_hash(key[b]);
        uuid[b] = get_uuid(key[b]);
        isDone[b] = false;
        isFound[b] = false;
      }
      #pragma unroll 2
      for (ll_t i=0; i < _numBuckets; i++){
        ll_t candidateUUID[BatchSize];
        #pragma unroll
        for (int b = 0; b < BatchSize; b++){
          address[b] = (hashCode[b] + i) % _numBuckets;
          candidateUUID[b] = get_uuid(address[b]);
        }
        #pragma unroll
        for (int b = 0; b < BatchSize; b++){
          // check if the candidateKey is emptyKey
          bool isEmpty = candidateUUID[b] == _emptyMarker;
          // is so, return not found
          if (isEmpty){
            isDone[b] = true;
          }
          // check if the candidateKey is equal to key
          isFound[b] = candidateUUID[b] == uuid[b];
          // if so, return found
          if (isFound[b]){
            get_value(address[b], value[b]);
            // return true;
            isDone[b] = true;
          }
        }
        bool isAllDone = isDone[0];
        #pragma unroll
        for (int b=1; b < BatchSize; b++){
          isAllDone = isAllDone && isDone[b];
        }
        if (isAllDone){
          break;
        }
      }
      #pragma unroll
      for (int b=0; b<BatchSize; b++){
        if (!isFound[b]){
          #pragma unroll
          for (int j=0; j<ValueSize; j++){
            value[b][j] = fallbackValue[j];
          }
        }
      }
    }

    template <int BatchSize>
    CUDA_DEVICE_INLINE
    void set_batched(
      KeyType key[BatchSize][KeySize],
      ValueType value[BatchSize][ValueSize],
      bool isStored[BatchSize]
    ){
      ll_t hashCode[BatchSize];
      ll_t uuid[BatchSize];
      bool isDone[BatchSize];
      #pragma unroll
      for (int b=0; b<BatchSize; b++){
        hashCode[b] = get_hash(key[b]);
        uuid[b] = get_uuid(key[b]);
        isDone[b] = false;
        isStored[b] = false;
      }

      #pragma unroll 2
      for (ll_t i=0; i<_numBuckets; i++){
        ll_t address[BatchSize];
        ll_t candidateUUID[BatchSize];
        bool isSuccessful[BatchSize];
        #pragma unroll
        for (int b=0; b<BatchSize; b++){
          address[b] = (hashCode[b] + i) % _numBuckets;
          isSuccessful[b] = set_uuid_if_empty(address[b], uuid[b], candidateUUID[b]);
        }

        #pragma unroll
        for (int b=0; b<BatchSize; b++){
          isStored[b] = isSuccessful[b] || (uuid[b] == candidateUUID[b]);
          if (isStored[b]){
            set_key(address[b], key[b]);
            set_value(address[b], value[b]);
            isDone[b] = true;
          }
        }

        bool isAllDone = isDone[0];
        #pragma unroll
        for (int b=1; b<BatchSize; b++){
          isAllDone = isAllDone && isDone[b];
        }
        if (isAllDone){
          break;
        }
      }
    }

    // CUDA_DEVICE_INLINE
    // void permute_key(KeyType key[KeySize]){
    //   KeyType permutedKey[KeySize];
    //   #pragma unroll
    //   for (int i=0; i<KeySize; i++){
    //     permutedKey[i] = key[keyPerm[i]];
    //   }
    //   #pragma unroll
    //   for (int i=0; i<KeySize; i++){
    //     key[i] = permutedKey[i];
    //   }
    // }
};


using KeyType = _KEYTYPE_;
using ValueType = _VALUETYPE_;
using BoolType = uint8_t;

extern "C"
__global__ void closed_hashmap_get(
  const ll_t* __restrict__ pPrime1, //[KeySize]
  const ll_t* __restrict__ pPrime2, //[KeySize]
  const ll_t* __restrict__ pAlpha1, //[KeySize]
  const ll_t* __restrict__ pAlpha2, //[KeySize]
  const ll_t* __restrict__ pBeta1,  //[KeySize]
  const ll_t* __restrict__ pBeta2,  //[KeySize]
  const ll_t* __restrict__ pKeyPerm,             //[KeySize]
  const KeyType* __restrict__ pKeys,             //[NumKeys, KeySize]
  ValueType* pValues,         //[NumKeys, ValueSize]
  KeyType* pAllKeys,          //[NumBuckets, KeySize]
  ValueType* pAllValues,      //[NumBuckets, ValueSize]
  ll_t* pAllUUIDs,            //[NumBuckets]
  const ValueType* __restrict__ pFallbackValue,  //[ValueSize]  
  BoolType* pIsFound,        //[NumKeys]
  ll_t numKeys, ll_t numBuckets
){
  constexpr int TPB = _TPB_;
  constexpr int KPT = _KPT_;
  constexpr int KeySize = _KEYSIZE_;
  constexpr int ValueSize = _VALUESIZE_;
  constexpr int KPB = TPB * KPT;

  int tid = threadIdx.x;
  ll_t kStart = blockIdx.x * KPB;

  ClosedHashmap<KeyType, ValueType, KeySize, ValueSize> hashmap(
    pPrime1, pPrime2,
    pAlpha1, pAlpha2,
    pBeta1,  pBeta2,
    pKeyPerm,
    pAllKeys,
    pAllValues,
    pAllUUIDs,
    numBuckets,
    -1
  );

  // Load keys
  KeyType keys[KPT][KeySize];
  ValueType values[KPT][ValueSize];
  ValueType fallbackValue[ValueSize];
  #pragma unroll
  for (int i=0; i<KPT; i++){
    ll_t offset = kStart + i * TPB + tid;
    if (offset < numKeys){
      #pragma unroll
      for (int j=0; j<KeySize; j++){
        keys[i][j] = pKeys[offset * KeySize + j];
        // keys[i][j] = pKeys[offset * KeySize + hashmap.keyPerm[j]];
      }
    }
  }
  
  #pragma unroll
  for (int i=0; i<ValueSize; i++){
    fallbackValue[i] = pFallbackValue[i];
  }

  // get values
  bool isFound[KPT];
  // hashmap.get_batched<KPT>(keys, values, fallbackValue, isFound);
  #pragma unroll
  for (int i=0; i<KPT; i++){
    int offset = kStart + i * TPB + tid;
    if (offset < numKeys){
      isFound[i] = hashmap.get(keys[i], values[i], fallbackValue);

      pIsFound[offset] = (BoolType) isFound[i];
      if (isFound[i]){
        #pragma unroll
        for (int j=0; j<ValueSize; j++){
          pValues[offset * ValueSize + j] = values[i][j];
        }
      }
    }
  }
}

extern "C"
__global__ void closed_hashmap_set(
  const ll_t* __restrict__ pPrime1, //[KeySize]
  const ll_t* __restrict__ pPrime2, //[KeySize]
  const ll_t* __restrict__ pAlpha1, //[KeySize]
  const ll_t* __restrict__ pAlpha2, //[KeySize]
  const ll_t* __restrict__ pBeta1,  //[KeySize]
  const ll_t* __restrict__ pBeta2,  //[KeySize]
  const ll_t* __restrict__ pKeyPerm,             //[KeySize]
  const KeyType* __restrict__ pKeys,             //[NumKeys, KeySize]
  const ValueType* __restrict__ pValues,         //[NumKeys, ValueSize]
  KeyType* pAllKeys,          //[NumBuckets, KeySize]
  ValueType* pAllValues,      //[NumBuckets, ValueSize]
  ll_t* pAllUUIDs,            //[NumBuckets]
  BoolType* pIsStored,        //[NumKeys]
  ll_t numKeys, ll_t numBuckets
){
  constexpr int TPB = _TPB_;
  constexpr int KPT = _KPT_;
  constexpr int KeySize = _KEYSIZE_;
  constexpr int ValueSize = _VALUESIZE_;
  constexpr int KPB = TPB * KPT;

  int tid = threadIdx.x;
  ll_t kStart = blockIdx.x * KPB;

  ClosedHashmap<KeyType, ValueType, KeySize, ValueSize> hashmap(
    pPrime1, pPrime2,
    pAlpha1, pAlpha2,
    pBeta1,  pBeta2,
    pKeyPerm,
    pAllKeys,
    pAllValues,
    pAllUUIDs,
    numBuckets,
    -1
  );

  // Load keys
  KeyType keys[KPT][KeySize];
  ValueType values[KPT][ValueSize];
  #pragma unroll
  for (int i=0; i<KPT; i++){
    ll_t offset = kStart + i * TPB + tid;
    if (offset < numKeys){
      #pragma unroll
      for (int j=0; j<KeySize; j++){
        keys[i][j] = pKeys[offset * KeySize + j];      
      }
      #pragma unroll
      for (int j=0; j<ValueSize; j++){
        values[i][j] = pValues[offset * ValueSize + j];
      }
    }
  }

  // get values
  bool isStored[KPT];
  // hashmap.set_batched<KPT>(keys, values, isStored);

  #pragma unroll
  for (int i=0; i<KPT; i++){
    int offset = kStart + i * TPB + tid;
    if (offset < numKeys){
      isStored[i] = hashmap.set(keys[i], values[i]);
      pIsStored[offset] = (BoolType) isStored[i];
    }
  }
}

// extern "C"
// __global__ void closed_hashmap_count_existing(
//   const ll_t* __restrict__ pPrime1, //[KeySize]
//   const ll_t* __restrict__ pPrime2, //[KeySize]
//   const ll_t* __restrict__ pAlpha1, //[KeySize]
//   const ll_t* __restrict__ pAlpha2, //[KeySize]
//   const ll_t* __restrict__ pBeta1,  //[KeySize]
//   const ll_t* __restrict__ pBeta2,  //[KeySize]
//   const KeyType* __restrict__ pKeys,             //[NumKeys, KeySize]
//   KeyType* pAllKeys,          //[NumBuckets, KeySize]
//   ValueType* pAllValues,      //[NumBuckets, ValueSize]
//   ll_t* pAllUUIDs,            //[NumBuckets]
//   ull_t* __restrict__ pCounts, //[1]
//   ll_t numKeys, ll_t numBuckets
// ){
//   constexpr int TPB = _TPB_;
//   constexpr int KPT = _KPT_;
//   constexpr int KeySize = _KEYSIZE_;
//   constexpr int ValueSize = _VALUESIZE_;
//   constexpr int KPB = TPB * KPT;

//   int tid = threadIdx.x;
//   ll_t kStart = blockIdx.x * KPB;

//   ClosedHashmap<KeyType, ValueType, KeySize, ValueSize> hashmap(
//     pPrime1, pPrime2,
//     pAlpha1, pAlpha2,
//     pBeta1,  pBeta2,
//     pAllKeys,
//     pAllValues,
//     pAllUUIDs,
//     numBuckets,
//     -1
//   );

//   // Load keys
//   KeyType keys[KPT][KeySize];
//   ValueType values[KPT][ValueSize];
//   ValueType fallbackValue[ValueSize];
//   #pragma unroll
//   for (int i=0; i<KPT; i++){
//     ll_t offset = kStart + i * TPB + tid;
//     if (offset < numKeys){
//       #pragma unroll
//       for (int j=0; j<KeySize; j++){
//         keys[i][j] = pKeys[offset * KeySize + j];
//       }
//     }
//   }
  
//   // #pragma unroll
//   // for (int i=0; i<ValueSize; i++){
//   //   fallbackValue[i] = pFallbackValue[i];
//   // }
//   __shared__ int blockCount[1];
//   // get values
//   int threadCount = 0;
//   // bool isFound[KPT];
//   #pragma unroll
//   for (int i=0; i<KPT; i++){
//     int offset = kStart + i * TPB + tid;
//     if (offset < numKeys){
//       bool isFound = hashmap.get(keys[i], values[i], fallbackValue);
//       if (isFound){
//         threadCount ++;
//       }
//     }
//   }

//   atomicAdd(blockCount, threadCount);
//   if (tid == 0){
//     atomicAdd(pCounts, (ull_t) blockCount[0]);
//   }
// }

// extern "C"
// __global__ void closed_hashmap_get_sparse(
//   const ll_t* __restrict__ pPrime1, //[KeySize]
//   const ll_t* __restrict__ pPrime2, //[KeySize]
//   const ll_t* __restrict__ pAlpha1, //[KeySize]
//   const ll_t* __restrict__ pAlpha2, //[KeySize]
//   const ll_t* __restrict__ pBeta1,  //[KeySize]
//   const ll_t* __restrict__ pBeta2,  //[KeySize]
//   const KeyType* __restrict__ pKeys,             //[NumKeys, KeySize]
//   KeyType* pAllKeys,          //[NumBuckets, KeySize]
//   ValueType* pAllValues,      //[NumBuckets, ValueSize]
//   ll_t* pAllUUIDs,            //[NumBuckets] 
  
//   const ll_t* __restrict__ pOutPrime1, //[KeySize]
//   const ll_t* __restrict__ pOutPrime2, //[KeySize]
//   const ll_t* __restrict__ pOutAlpha1, //[KeySize]
//   const ll_t* __restrict__ pOutAlpha2, //[KeySize]
//   const ll_t* __restrict__ pOutBeta1,  //[KeySize]
//   const ll_t* __restrict__ pOutBeta2,  //[KeySize]
//   KeyType* pOutAllKeys,          //[NumBuckets, KeySize]
//   ValueType* pOutAllValues,      //[NumBuckets, ValueSize]
//   ll_t* pOutAllUUIDs,            //[NumBuckets] 

//   ll_t numKeys, ll_t numBuckets, ll_t numOutBuckets
// ){
//   constexpr int TPB = _TPB_;
//   constexpr int KPT = _KPT_;
//   constexpr int KeySize = _KEYSIZE_;
//   constexpr int ValueSize = _VALUESIZE_;
//   constexpr int KPB = TPB * KPT;

//   int tid = threadIdx.x;
//   ll_t kStart = blockIdx.x * KPB;

//   ClosedHashmap<KeyType, ValueType, KeySize, ValueSize> hashmap(
//     pPrime1, pPrime2,
//     pAlpha1, pAlpha2,
//     pBeta1,  pBeta2,
//     pAllKeys,
//     pAllValues,
//     pAllUUIDs,
//     numBuckets,
//     -1
//   );

//   ClosedHashmap<KeyType, ValueType, KeySize, ValueSize> outHashmap(
//     pOutPrime1, pOutPrime2,
//     pOutAlpha1, pOutAlpha2,
//     pOutBeta1,  pOutBeta2,
//     pOutAllKeys,
//     pOutAllValues,
//     pOutAllUUIDs,
//     numOutBuckets,
//     -1
//   );


//   // Load keys
//   KeyType keys[KPT][KeySize];
//   ValueType values[KPT][ValueSize];
//   ValueType fallbackValue[ValueSize];
//   #pragma unroll
//   for (int i=0; i<KPT; i++){
//     ll_t offset = kStart + i * TPB + tid;
//     if (offset < numKeys){
//       #pragma unroll
//       for (int j=0; j<KeySize; j++){
//         keys[i][j] = pKeys[offset * KeySize + j];
//       }
//     }
//   }
  
//   #pragma unroll
//   for (int i=0; i<ValueSize; i++){
//     fallbackValue[i] = pFallbackValue[i];
//   }

//   // get values
//   bool isFound[KPT];
//   // hashmap.get_batched<KPT>(keys, values, fallbackValue, isFound);
//   #pragma unroll
//   for (int i=0; i<KPT; i++){
//     int offset = kStart + i * TPB + tid;
//     if (offset < numKeys){
//       isFound[i] = hashmap.get(keys[i], values[i], fallbackValue);
//       if (isFound[i]){
//         outHashmap.set(keys[i], values[i]);
//       //   #pragma unroll
//       //   for (int j=0; j<ValueSize; j++){
//       //     pValues[offset * ValueSize + j] = values[i][j];
//       //   }
//       }
//     }
//   }
//   #pragma unroll
//   for (int i=0; i<KPT; i++){
//     int offset = kStart + i * TPB + tid;
//     if (offset < numKeys){
// }