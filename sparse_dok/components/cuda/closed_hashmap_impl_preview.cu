#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"
#include <stdio.h>
// #include "mma"

#define likely(x)      __builtin_expect(!!(x), 1)
#define unlikely(x)    __builtin_expect(!!(x), 0)
#define load(x)        __ldcg(x)
#define store(x, value) __stcs(x, value)
#define div_ru(a, b) (a + b - 1) / b 
#define div_rd(a, b) a / b 
#define VOLATILE
#ifndef INFINITY
#define INFINITY __int_as_float(0x7f800000)
#endif
#define DEBUG

#ifdef DEBUG
#define COMPILER_ASSERT(EXPRESSION)   switch (0) {case 0: case (EXPRESSION):;}
#else
#define COMPILER_ASSERT(EXPRESSION)
#endif

#define CUDA_DEVICE_INLINE __device__ __forceinline__

// #define LAYOUT_C true
// #define LAYOUT_F false
// #define TRANSFORM_N true
// #define TRANSFORM_T false

// typedef bool MATRIX_LAYOUT;
// typedef bool MATRIX_TRANSFORM;
typedef unsigned char uint8_t;
typedef long long ll_t;
typedef unsigned long long ull_t;

typedef struct __builtin_align__(8){
  half x1, x2, x3, x4;
} half4;

typedef struct __builtin_align__(16){
  half x1, x2, x3, x4, x5, x6, x7, x8;
} half8;

typedef struct __builtin_align__(16){
  unsigned char x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15, x16;
} uchar16;

typedef struct __builtin_align__(16){
  char x1, x2, x3, x4, x5, x6, x7, x8, x9, x10, x11, x12, x13, x14, x15, x16;
} char16;

typedef struct {
  int x;
} Coord1D;

typedef struct {
  int x, y;
} Coord2D;

typedef struct {
  int x, y, z;
} Coord3D;

typedef struct {
  int x, y, z, t;
} Coord4D;

typedef struct {
  int x, y, z, t, u;
} Coord5D;

typedef union {
  int as_int32[1];
  unsigned int as_uint32[1];
  short as_int16[2];
  unsigned short as_uint16[2];
  signed char as_int8[4];
  unsigned char as_uint8[4]; 
  float as_float[1];
  half2 as_half2[1];
  half as_half[2];  
} Data4B;

typedef union {
  long long as_int64[1];
  unsigned long long as_uint64[1];
  int as_int32[2];
  unsigned int as_uint32[2];
  short as_int16[4];
  unsigned short as_uint16[4];
  signed char as_int8[8];
  unsigned char as_uint8[8]; 
  double as_double[1];
  half4 as_half4[1];
  float2 as_float2[1];
  float as_float[2];
  half2 as_half2[2];
  half as_half[4];  
} Data8B;

typedef union {
  uchar16 as_uchar16[1];
  char16 as_char16[1];
  long long as_int64[2];
  unsigned long long as_uint64[2];
  int as_int32[4];
  unsigned int as_uint32[4];
  short as_int16[8];
  unsigned short as_uint16[8];
  signed char as_int8[16];
  unsigned char as_uint8[16];
  half8 as_half8[1]; 
  double as_double[2];
  half4 as_half4[2];
  float2 as_float2[2];
  float as_float[4];
  half2 as_half2[4];
  half as_half[8];  
} Data16B;


template <typename MutexType>
CUDA_DEVICE_INLINE
void mutex_lock_thread(
  MutexType *mutex,
  const MutexType onValue,
  const MutexType offValue
) {
  unsigned int ns = 8;
  unsigned int counter = 0;
  while (atomicCAS(mutex, offValue, onValue) == onValue) {
    __nanosleep(ns);
    counter ++;
    if (counter > 1000) break;
    if (ns < 256) {
      ns *= 2;
    }
  }
}

template <typename MutexType>
CUDA_DEVICE_INLINE
void mutex_unlock_thread(
  MutexType *mutex,
  const MutexType offValue
) {
  __threadfence();
  atomicExch(mutex, offValue);
  __threadfence();
}

CUDA_DEVICE_INLINE
long long atomicCAS(
  ll_t *address,
  ll_t compare,
  ll_t val
){
  ull_t old = atomicCAS(
    reinterpret_cast<ull_t*>(address),
    reinterpret_cast<ull_t&>(compare),
    reinterpret_cast<ull_t&>(val)
  );
  return reinterpret_cast<ll_t&>(old);
}

CUDA_DEVICE_INLINE
long long atomicExch(
  ll_t *address,
  ll_t val
){
  ull_t old = atomicExch(
    reinterpret_cast<ull_t*>(address),
    reinterpret_cast<ull_t&>(val)
  );
  return reinterpret_cast<ll_t&>(old);
}

CUDA_DEVICE_INLINE
ll_t atomicAdd(
  ll_t *address,
  ll_t val
){
  ull_t old = atomicAdd(
    reinterpret_cast<ull_t*>(address),
    reinterpret_cast<ull_t&>(val)
  );
  return reinterpret_cast<ll_t&>(old);
}
template <
  typename T
>
class SmemTensor0D{
  public:
    VOLATILE T* endPtr;
    VOLATILE T* startPtr;

    CUDA_DEVICE_INLINE
    SmemTensor0D(VOLATILE void* smemPtr)  
        : startPtr(reinterpret_cast<T*>(smemPtr))
        , endPtr(reinterpret_cast<T*>(smemPtr))
    {
    }

    CUDA_DEVICE_INLINE
    T get(){
      return startPtr[0];
    }

    CUDA_DEVICE_INLINE
    T* get_ptr(){
      return startPtr;
    }

    CUDA_DEVICE_INLINE
    void set(T value){
      startPtr[0] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U get_reinterpreted(){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return newPtr[0];
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    void set_reinterpreted(U value){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      newPtr[0] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U* get_ptr_reinterpreted(){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return &newPtr[0];
    }
};

template <
  typename T,
  int ShapeX
>
class SmemTensor1D{
  public:
    VOLATILE T* endPtr;
    VOLATILE T* startPtr;

    CUDA_DEVICE_INLINE
    SmemTensor1D(VOLATILE void* smemPtr)  
        : startPtr(reinterpret_cast<T*>(smemPtr))
        , endPtr(reinterpret_cast<T*>(smemPtr) + shape().x)
    {
    }

    CUDA_DEVICE_INLINE
    T get(int x){
      return startPtr[x];
    }

    CUDA_DEVICE_INLINE
    T* get_ptr(int x){
      return &startPtr[x];
    }

    CUDA_DEVICE_INLINE
    void set(int x, T value){
      startPtr[x] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U get_reinterpreted(int x){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return newPtr[x];
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    void set_reinterpreted(int x, U value){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      newPtr[x] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U* get_ptr_reinterpreted(int x){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return &newPtr[x];
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord1D shape(){
      return { ShapeX };
    }
};

template <
  typename T,
  int ShapeX,
  int ShapeY
>
class SmemTensor2D{
  public:
    VOLATILE T* endPtr;
    VOLATILE T* startPtr;

    CUDA_DEVICE_INLINE
    SmemTensor2D(VOLATILE void* smemPtr)  
        : startPtr(reinterpret_cast<T*>(smemPtr))
        , endPtr(reinterpret_cast<T*>(smemPtr) + shape().x * shape().y)
    {
    }

    CUDA_DEVICE_INLINE
    T get(int x, int y){
      return startPtr[x * stride().x + y];
    }

    CUDA_DEVICE_INLINE
    T* get_ptr(int x, int y){
      return &startPtr[x * stride().x + y];
    }

    CUDA_DEVICE_INLINE
    void set(int x, int y, T value){
      startPtr[x * stride().x + y] = value;
    }

    CUDA_DEVICE_INLINE
    SmemTensor1D<T, ShapeY> get_child(int x){
      SmemTensor1D<T, ShapeY> child(
        &startPtr[x * stride().x]
      );
      return child;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U get_reinterpreted(int x, int y){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return newPtr[
        (x * stride().x) * sizeof(T) / sizeof(U) + 
        y
      ];
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    void set_reinterpreted(int x, int y, U value){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      newPtr[
        (x * stride().x) * sizeof(T) / sizeof(U) + 
        y
      ] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U* get_ptr_reinterpreted(int x, int y){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return &newPtr[
        (x * stride().x) * sizeof(T) / sizeof(U) + 
        y
      ];
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord2D shape(){
      return {ShapeX, ShapeY};
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord1D stride(){
      return {ShapeY};
    }

};

template <
  typename T,
  int ShapeX,
  int ShapeY,
  int ShapeZ
>
class SmemTensor3D{
  public:
    VOLATILE T* endPtr;
    VOLATILE T* startPtr;

    CUDA_DEVICE_INLINE
    SmemTensor3D(VOLATILE void* smemPtr)  
        : startPtr(reinterpret_cast<T*>(smemPtr))
        , endPtr(reinterpret_cast<T*>(smemPtr) + shape().x * shape().y * shape().z)
    {
    }

    CUDA_DEVICE_INLINE
    T get(int x, int y, int z){
      return startPtr[x * stride().x + y * stride().y + z];
    }

    CUDA_DEVICE_INLINE
    T* get_ptr(int x, int y, int z){
      return &startPtr[x * stride().x + y * stride().y + z];
    }

    CUDA_DEVICE_INLINE
    void set(int x, int y, int z, T value){
      startPtr[x * stride().x + y * stride().y + z] = value;
    }

    CUDA_DEVICE_INLINE
    SmemTensor2D<T, ShapeY, ShapeZ> get_child(int x){
      SmemTensor2D<T, ShapeY, ShapeZ> child(
        &startPtr[x * stride().x]
      );
      return child;
    }

    CUDA_DEVICE_INLINE
    SmemTensor1D<T, ShapeZ> get_child(int x, int y){
      SmemTensor1D<T, ShapeZ> child(
        &startPtr[x * stride().x + y * stride().y]
      );
      return child;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U get_reinterpreted(int x, int y, int z){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return newPtr[
        (x * stride().x +  
        y * stride().y) * sizeof(T) / sizeof(U) + 
        z
      ];
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    void set_reinterpreted(int x, int y, int z, U value){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      newPtr[
        (x * stride().x +  
        y * stride().y) * sizeof(T) / sizeof(U) + 
        z
      ] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U* get_ptr_reinterpreted(int x, int y, int z){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return &newPtr[
        (x * stride().x +  
        y * stride().y) * sizeof(T) / sizeof(U) + 
        z
      ];
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord3D shape(){
      return {ShapeX, ShapeY, ShapeZ};
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord2D stride(){
      return {ShapeY * ShapeZ, ShapeZ};
    }

};

template <
  typename T,
  int ShapeX,
  int ShapeY,
  int ShapeZ,
  int ShapeT
>
class SmemTensor4D{
  public:
    VOLATILE T* endPtr;
    VOLATILE T* startPtr;
    // const Coord3D _stride;
    // const Coord4D _shape;

    CUDA_DEVICE_INLINE
    SmemTensor4D(VOLATILE void* smemPtr)  
        : startPtr(reinterpret_cast<T*>(smemPtr))
        , endPtr(&reinterpret_cast<T*>(smemPtr)[shape().x * shape().y * shape().z * shape().t])
    {
    }

    CUDA_DEVICE_INLINE
    T get(int x, int y, int z, int t){
      return startPtr[
        x * stride().x + 
        y * stride().y + 
        z * stride().z +
        t
      ];
    }

    CUDA_DEVICE_INLINE
    T* get_ptr(int x, int y, int z, int t){
      return &startPtr[
        x * stride().x + 
        y * stride().y + 
        z * stride().z +
        t
      ];
    }

    CUDA_DEVICE_INLINE
    void set(int x, int y, int z, int t, T value){
      startPtr[
        x * stride().x + 
        y * stride().y + 
        z * stride().z +
        t
      ] = value;
    }

    CUDA_DEVICE_INLINE
    SmemTensor3D<T, ShapeY, ShapeZ, ShapeT> get_child(int x){
      SmemTensor3D<T, ShapeY, ShapeZ, ShapeT> child(
        &startPtr[x * stride().x]
      );
      return child;
    }

    CUDA_DEVICE_INLINE
    SmemTensor2D<T, ShapeZ, ShapeT> get_child(int x, int y){
      SmemTensor2D<T, ShapeZ, ShapeT> child(
        &startPtr[x * stride().x + y * stride().y]
      );
      return child;
    }

    CUDA_DEVICE_INLINE
    SmemTensor1D<T, ShapeT> get_child(int x, int y, int z){
      SmemTensor1D<T, ShapeT> child(
        &startPtr[x * stride().x + y * stride().y + z * stride().z]
      );
      return child;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U get_reinterpreted(int x, int y, int z, int t){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return newPtr[
        (x * stride().x +  
        y * stride().y +  
        z * stride().z) * sizeof(T) / sizeof(U) + 
        t
      ];
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    void set_reinterpreted(int x, int y, int z, int t, U value){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      newPtr[
        (x * stride().x +  
        y * stride().y +  
        z * stride().z) * sizeof(T) / sizeof(U) + 
        t
      ] = value;
    }

    template <typename U>
    CUDA_DEVICE_INLINE
    U* get_ptr_reinterpreted(int x, int y, int z, int t){
      U* newPtr = reinterpret_cast<U*>(startPtr);
      return &newPtr[
        (x * stride().x +  
        y * stride().y +  
        z * stride().z) * sizeof(T) / sizeof(U) + 
        t
      ];
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord4D shape(){
      return {ShapeX, ShapeY, ShapeZ, ShapeT};
    }

    CUDA_DEVICE_INLINE
    static constexpr Coord3D stride(){
      return {
        ShapeY * ShapeZ * ShapeT, 
        ShapeZ * ShapeT, 
        ShapeT
      };
    }
};
#define EMPTY 1
#define FOUND 2
#define NOT_FOUND 3
#define STORED 4
#define NOT_STORED 5

template <
  typename key_t,
  typename value_t,
  int KeySize,
  int ValueSize
>
class ClosedHashmap{
  private:
    ll_t _prime1[KeySize];
    ll_t _prime2[KeySize];
    ll_t _alpha1[KeySize];
    ll_t _alpha2[KeySize];
    ll_t _beta1[KeySize];
    ll_t _beta2[KeySize];
    key_t *_pAllKeys;
    value_t *_pAllValues;
    ll_t *_pAllUUIDs;
    ll_t _numBuckets;
    // ll_t _numElements;
    ll_t _emptyMarker;
    ll_t _removedMarker;

  public:
    ll_t keyPerm[KeySize];

    CUDA_DEVICE_INLINE
    ClosedHashmap(const ll_t* pPrime1,
                  const ll_t* pPrime2,
                  const ll_t* pAlpha1,
                  const ll_t* pAlpha2,
                  const ll_t* pBeta1,
                  const ll_t* pBeta2,
                  const ll_t* pKeyPerm,
                  key_t* pAllKeys,
                  value_t* pAllValues,
                  ll_t* pAllUUIDs,
                  ll_t numBuckets,
                  ll_t emptyMarker,
                  ll_t removedMarker
                  )
                  : _pAllKeys(pAllKeys)
                  , _pAllValues(pAllValues)
                  , _pAllUUIDs(pAllUUIDs)
                  , _numBuckets(numBuckets)
                  , _emptyMarker(emptyMarker)
                  , _removedMarker(removedMarker)
    {
      #pragma unroll
      for (int i=0; i < KeySize; i++){
        keyPerm[i] = pKeyPerm[i];
        _prime1[i] = pPrime1[keyPerm[i]];
        _prime2[i] = pPrime2[keyPerm[i]];
        _alpha1[i] = pAlpha1[keyPerm[i]];
        _alpha2[i] = pAlpha2[keyPerm[i]];
        _beta1[i] = pBeta1[keyPerm[i]];
        _beta2[i] = pBeta2[keyPerm[i]];
      }
    }

    CUDA_DEVICE_INLINE
    ClosedHashmap(const ll_t* pArgs){
      #pragma unroll
      for (int i=0; i < KeySize; i++){
        _prime1[i] = pArgs[i];
        _prime2[i] = pArgs[i + KeySize * 1];
        _alpha1[i] = pArgs[i + KeySize * 2];
        _alpha2[i] = pArgs[i + KeySize * 3];
        _beta1[i] = pArgs[i + KeySize * 4];
        _beta2[i] = pArgs[i + KeySize * 5];
        keyPerm[i] = pArgs[i + KeySize * 6];
      }
      _pAllKeys = reinterpret_cast<key_t*>(pArgs[KeySize * 7]);
      _pAllValues = reinterpret_cast<value_t*>(pArgs[KeySize * 7 + 1]);
      _pAllUUIDs = reinterpret_cast<ll_t*>(pArgs[KeySize * 7 + 2]);
      _numBuckets = pArgs[KeySize * 7 + 3];
      // _numElements = pArgs[KeySize * 7 + 4]; //FIXME: useless for now
      _emptyMarker = pArgs[KeySize * 7 + 4];
      _removedMarker = pArgs[KeySize * 7 + 5];
    }

    CUDA_DEVICE_INLINE
    ll_t get_hash(key_t key[KeySize]){
      ll_t hash_code = ( (ll_t) key[0] * _alpha1[0] + _beta1[0]) % _prime1[0];
      #pragma unroll
      for (int i=1; i<KeySize; i++){
        hash_code *= ( (ll_t) key[i] * _alpha1[i] + _beta1[i]) % _prime1[i];
      }
      hash_code = llabs(hash_code);
      return hash_code;
    }

    CUDA_DEVICE_INLINE
    ll_t get_uuid(key_t key[KeySize]){
      ll_t uuid = ( (ll_t) key[0] * _alpha2[0] + _beta2[0]) % _prime2[0];
      #pragma unroll
      for (int i=1; i<KeySize; i++){
        uuid *= ( (ll_t) key[i] * _alpha2[i] + _beta2[i]) % _prime2[i];
      }
      uuid = llabs(uuid);
      return uuid;
    }

    CUDA_DEVICE_INLINE
    bool are_keys_equal(key_t key1[KeySize], key_t key2[KeySize]){
      bool isEqual = key1[0] == key2[0];
      #pragma unroll
      for (int i=0; i<KeySize; i++){
        isEqual = isEqual && (key1[i] == key2[i]);
      }
      return isEqual;
    }

    CUDA_DEVICE_INLINE
    void get_key(ll_t address, key_t key[KeySize]){
      #pragma unroll
      for (int i=0; i<KeySize; i++){
        key[i] = _pAllKeys[address * KeySize + i];
      }
    }

    CUDA_DEVICE_INLINE
    void get_key_permuted(ll_t address, key_t key[KeySize]){
      #pragma unroll
      for (int i=0; i<KeySize; i++){
        key[i] = _pAllKeys[address * KeySize + keyPerm[i]];
      }
    }

    CUDA_DEVICE_INLINE
    void set_key(ll_t address, key_t key[KeySize]){
      #pragma unroll
      for (int i=0; i<KeySize; i++){
        _pAllKeys[address * KeySize + i] = key[i];
      }
    
    }

    CUDA_DEVICE_INLINE
    void set_key_permuted(ll_t address, key_t key[KeySize]){
      #pragma unroll
      for (int i=0; i<KeySize; i++){
        _pAllKeys[address * KeySize + keyPerm[i] ] = key[i];
      }
    }

    CUDA_DEVICE_INLINE
    void get_value(ll_t address, value_t value[ValueSize]){
      #pragma unroll
      for (int i=0; i<ValueSize; i++){
        value[i] = _pAllValues[address * ValueSize + i];
      }
    }

    CUDA_DEVICE_INLINE
    void set_value(ll_t address, value_t value[ValueSize]){
      #pragma unroll
      for (int i=0; i<ValueSize; i++){
        _pAllValues[address * ValueSize + i] = value[i];
      }
    }

    CUDA_DEVICE_INLINE
    void get_uuid(ll_t address, ll_t &uuid){
      uuid = _pAllUUIDs[address];
    }

    CUDA_DEVICE_INLINE
    ll_t get_uuid(ll_t address){
      return _pAllUUIDs[address];
    }

    CUDA_DEVICE_INLINE
    void set_uuid(ll_t address, ll_t uuid){
      _pAllUUIDs[address] = uuid;
    }

    CUDA_DEVICE_INLINE
    bool set_uuid_if_empty(int address, ll_t uuid, ll_t &oldUUID){
      ll_t *ptr = &_pAllUUIDs[address];
      // if the value at `ptr` is equal to `_emptyMarker`, then set the value of that pointer to `uuid`, return true
      // else, return false
      __threadfence();
      oldUUID = atomicCAS(ptr, _emptyMarker, uuid);
      __threadfence();
      if ( oldUUID == _emptyMarker){
        return true;
      }
      return false;
    }

    CUDA_DEVICE_INLINE
    bool set_uuid_if_removed(int address, ll_t uuid, ll_t &oldUUID){
      ll_t *ptr = &_pAllUUIDs[address];
      // if the value at `ptr` is equal to `_removedMarker`, then set the value of that pointer to `uuid`, return true
      // else, return false
      __threadfence();
      oldUUID = atomicCAS(ptr, _removedMarker, uuid);
      __threadfence();
      if ( oldUUID == _removedMarker){
        return true;
      }
      return false;
    }

    CUDA_DEVICE_INLINE
    int get_by_uuid(ll_t address, ll_t uuid, value_t value[ValueSize]){
      ll_t candidateUUID = get_uuid(address);
      // check if the candidateKey is emptyKey
      bool isEmpty = candidateUUID == _emptyMarker;
      // is so, return not found
      if (isEmpty){
        return EMPTY;
      }
      // check if the candidateKey is equal to key
      bool isFound = candidateUUID == uuid;
      // if so, return found
      if (isFound){
        get_value(address, value);
        return FOUND;
      }
      return NOT_FOUND;
    }

     CUDA_DEVICE_INLINE
    int set_by_uuid(int address, ll_t uuid, key_t key[KeySize], value_t value[ValueSize]){
      // is so, store key and value in this address
      // set key to that address, if storing failed (because of another thread using that address ), return not stored
      ll_t candidateUUID;
      bool isSuccessful = set_uuid_if_empty(address, uuid, candidateUUID);
      if (isSuccessful){
        set_key(address, key);
        set_value(address, value);
        return STORED;
      }
      // check if the candidateUUID is equal to uuid
      bool isFound = uuid == candidateUUID;
      // if so, return stored
      if (isFound){
        set_key(address, key);
        set_value(address, value);
        return STORED;
      }
      // otherwise, return not found
      return NOT_STORED;
    }

    CUDA_DEVICE_INLINE
    bool exists(
      key_t key[KeySize]
    ){
      // permute_key(key);
      ll_t hashCode = get_hash(key);
      ll_t uuid = get_uuid(key);
      #pragma unroll 2
      for (ll_t i=0; i < _numBuckets; i++){
        ll_t address = (hashCode + i) % _numBuckets;
        ll_t candidateUUID = get_uuid(address);
        // check if the candidateKey is emptyKey
        bool isEmpty = candidateUUID == _emptyMarker;
        // is so, return not found
        if (isEmpty){
          break;
        }
        // check if the candidateKey is equal to key
        bool isFound = candidateUUID == uuid;
        // if so, return found
        if (isFound){
          return true;
        }
      }
      return false;
    }

    CUDA_DEVICE_INLINE
    bool get(
      key_t key[KeySize],
      value_t value[ValueSize],
      value_t fallbackValue[ValueSize]
    ){
      // permute_key(key);
      ll_t hashCode = get_hash(key);
      ll_t uuid = get_uuid(key);
      #pragma unroll 2
      for (ll_t i=0; i < _numBuckets; i++){
        ll_t address = (hashCode + i) % _numBuckets;
        ll_t candidateUUID = get_uuid(address);
        // check if the candidateKey is emptyKey
        bool isEmpty = candidateUUID == _emptyMarker;
        // is so, return not found
        if (isEmpty){
          break;
        }
        // check if the candidateKey is equal to key
        bool isFound = candidateUUID == uuid;
        // if so, return found
        if (isFound){
          get_value(address, value);
          return true;
        }
      }
      #pragma unroll
      for (int j=0; j<ValueSize; j++){
        value[j] = fallbackValue[j];
      }
      return false;
    }

    CUDA_DEVICE_INLINE
    bool set_v0(
      key_t key[KeySize],
      value_t value[ValueSize]
    ){
      // permute_key(key);
      ll_t hashCode = get_hash(key);
      ll_t uuid = get_uuid(key);
      ll_t firstRemovedAddress = -1;
      #pragma unroll 2
      for (ll_t i=0; i<_numBuckets; i++){
        ll_t address = (hashCode + i) % _numBuckets;
        ll_t candidateUUID;
        candidateUUID = get_uuid(address);
        bool isFound = candidateUUID == uuid;
        // if key is found, return stored
        if (isFound){
          set_key_permuted(address, key);
          set_value(address, value);
          return true;
        }

        bool isRemoved = candidateUUID == _removedMarker;
        if (isRemoved && firstRemovedAddress == -1){
          firstRemovedAddress = address;
        }

        bool isEmpty = candidateUUID == _emptyMarker;
        if (isEmpty){
          // if no deletedMarker encountered previously, store key-value pair to nearest empty address.
          if (firstRemovedAddress == -1){
            bool isSuccessful = set_uuid_if_empty(address, uuid, candidateUUID);
            if (isSuccessful){
              set_key_permuted(address, key);
              set_value(address, value);
              return true;
            }
          } else {
          // otherwise, try to store the key-value pair to that deletedMarker, if fail, store to nearest empty address.
            bool isSuccessful = set_uuid_if_removed(firstRemovedAddress, uuid, candidateUUID);
            if (isSuccessful){
              set_key_permuted(firstRemovedAddress, key);
              set_value(firstRemovedAddress, value);
              return true;
            } else {
              firstRemovedAddress = -1;
              i--;
            }
          }
        }
      }
      return false;
    }

    CUDA_DEVICE_INLINE
    bool set(
      key_t key[KeySize],
      value_t value[ValueSize]
    ){
      // permute_key(key);
      ll_t startAddress = get_hash(key);
      ll_t uuid = get_uuid(key);
      ll_t firstRemovedAddress = -1;
      #pragma unroll 2
      for (ll_t i=0; i<_numBuckets; i++){
        ll_t address = (startAddress + i) % _numBuckets;
        ll_t candidateUUID;
        candidateUUID = get_uuid(address);
        bool isFound = candidateUUID == uuid;
        // if key is found, return stored
        if (isFound){
          set_key_permuted(address, key);
          set_value(address, value);
          return true;
        }

        bool isRemoved = candidateUUID == _removedMarker;
        if (isRemoved && firstRemovedAddress == -1){
          firstRemovedAddress = address;
        }

        bool isEmpty = candidateUUID == _emptyMarker;
        if (isEmpty){
          if (firstRemovedAddress == -1){
            if (set_uuid_if_empty(address, uuid, candidateUUID)){
              set_key_permuted(address, key);
              set_value(address, value);
              return true;
            }
          } else {
            break;
          }
        }
      }

      if (firstRemovedAddress != -1){
        #pragma unroll 2
        for (ll_t i=0; i<_numBuckets; i++){
          ll_t address = (firstRemovedAddress + i) % _numBuckets;
          ll_t candidateUUID;
          // candidateUUID = get_uuid(address);
          if (set_uuid_if_removed(address, uuid, candidateUUID)){
            set_key_permuted(address, key);
            set_value(address, value);
            return true;
          } else if (set_uuid_if_empty(address, uuid, candidateUUID)){
            set_key_permuted(address, key);
            set_value(address, value);
            return true;
          }
        }
      }
      return false;
    }

    CUDA_DEVICE_INLINE
    bool set_old(
      key_t key[KeySize],
      value_t value[ValueSize]
    ){
      // permute_key(key);
      ll_t hashCode = get_hash(key);
      ll_t uuid = get_uuid(key);
      #pragma unroll 2
      for (ll_t i=0; i<_numBuckets; i++){
        ll_t address = (hashCode + i) % _numBuckets;
        ll_t candidateUUID;
        bool isSuccessful = set_uuid_if_empty(address, uuid, candidateUUID);
        if (isSuccessful){
          set_key_permuted(address, key);
          set_value(address, value);
          return true;
        }
        // check if the candidateUUID is equal to uuid
        bool isFound = uuid == candidateUUID;
        // if so, return stored
        if (isFound){
          set_key_permuted(address, key);
          set_value(address, value);
          return true;
        }
      }
      return false;
    }

    CUDA_DEVICE_INLINE
    bool remove(
      key_t key[KeySize]
    ){
      ll_t hashCode = get_hash(key);
      ll_t uuid = get_uuid(key);
      #pragma unroll 2
      for (ll_t i=0; i < _numBuckets; i++){
        ll_t address = (hashCode + i) % _numBuckets;
        ll_t candidateUUID = get_uuid(address);
        // check if the candidateKey is emptyKey
        bool isEmpty = candidateUUID == _emptyMarker;
        // is so, return not found
        if (isEmpty){
          break;
        }
        // check if the candidateKey is equal to key
        bool isFound = candidateUUID == uuid;
        // if so, return found
        if (isFound){
          set_uuid(address, _removedMarker);
          return true;
        }
      }
      return false;
    }

    template <int BatchSize>
    CUDA_DEVICE_INLINE
    void get_batched(
      key_t key[BatchSize][KeySize],
      value_t value[BatchSize][ValueSize],
      value_t fallbackValue[ValueSize],
      bool isFound[BatchSize]
    ){
      ll_t hashCode[BatchSize];
      ll_t uuid[BatchSize];
      bool isDone[BatchSize];
      ll_t address[BatchSize];
      #pragma unroll
      for (int b = 0; b < BatchSize; b++){
        hashCode[b] = get_hash(key[b]);
        uuid[b] = get_uuid(key[b]);
        isDone[b] = false;
        isFound[b] = false;
      }
      #pragma unroll 2
      for (ll_t i=0; i < _numBuckets; i++){
        ll_t candidateUUID[BatchSize];
        #pragma unroll
        for (int b = 0; b < BatchSize; b++){
          address[b] = (hashCode[b] + i) % _numBuckets;
          candidateUUID[b] = get_uuid(address[b]);
        }
        #pragma unroll
        for (int b = 0; b < BatchSize; b++){
          // check if the candidateKey is emptyKey
          bool isEmpty = candidateUUID[b] == _emptyMarker;
          // is so, return not found
          if (isEmpty){
            isDone[b] = true;
          }
          // check if the candidateKey is equal to key
          isFound[b] = candidateUUID[b] == uuid[b];
          // if so, return found
          if (isFound[b]){
            get_value(address[b], value[b]);
            // return true;
            isDone[b] = true;
          }
        }
        bool isAllDone = isDone[0];
        #pragma unroll
        for (int b=1; b < BatchSize; b++){
          isAllDone = isAllDone && isDone[b];
        }
        if (isAllDone){
          break;
        }
      }
      #pragma unroll
      for (int b=0; b<BatchSize; b++){
        if (!isFound[b]){
          #pragma unroll
          for (int j=0; j<ValueSize; j++){
            value[b][j] = fallbackValue[j];
          }
        }
      }
    }

    template <int BatchSize>
    CUDA_DEVICE_INLINE
    void set_batched(
      key_t key[BatchSize][KeySize],
      value_t value[BatchSize][ValueSize],
      bool isStored[BatchSize]
    ){
      ll_t hashCode[BatchSize];
      ll_t uuid[BatchSize];
      bool isDone[BatchSize];
      #pragma unroll
      for (int b=0; b<BatchSize; b++){
        hashCode[b] = get_hash(key[b]);
        uuid[b] = get_uuid(key[b]);
        isDone[b] = false;
        isStored[b] = false;
      }

      #pragma unroll 2
      for (ll_t i=0; i<_numBuckets; i++){
        ll_t address[BatchSize];
        ll_t candidateUUID[BatchSize];
        bool isSuccessful[BatchSize];
        #pragma unroll
        for (int b=0; b<BatchSize; b++){
          address[b] = (hashCode[b] + i) % _numBuckets;
          isSuccessful[b] = set_uuid_if_empty(address[b], uuid[b], candidateUUID[b]);
        }

        #pragma unroll
        for (int b=0; b<BatchSize; b++){
          isStored[b] = isSuccessful[b] || (uuid[b] == candidateUUID[b]);
          if (isStored[b]){
            set_key(address[b], key[b]);
            set_value(address[b], value[b]);
            isDone[b] = true;
          }
        }

        bool isAllDone = isDone[0];
        #pragma unroll
        for (int b=1; b<BatchSize; b++){
          isAllDone = isAllDone && isDone[b];
        }
        if (isAllDone){
          break;
        }
      }
    }
};

using KeyType = _KEYTYPE_;
using ValueType = _VALUETYPE_;
using BoolType = uint8_t;

extern "C"
__global__ void closed_hashmap_get(
  const ll_t* __restrict__ pPrime1, //[KeySize]
  const ll_t* __restrict__ pPrime2, //[KeySize]
  const ll_t* __restrict__ pAlpha1, //[KeySize]
  const ll_t* __restrict__ pAlpha2, //[KeySize]
  const ll_t* __restrict__ pBeta1,  //[KeySize]
  const ll_t* __restrict__ pBeta2,  //[KeySize]
  const ll_t* __restrict__ pKeyPerm,             //[KeySize]
  const KeyType* __restrict__ pKeys,             //[NumKeys, KeySize]
  ValueType* pValues,         //[NumKeys, ValueSize]
  KeyType* pAllKeys,          //[NumBuckets, KeySize]
  ValueType* pAllValues,      //[NumBuckets, ValueSize]
  ll_t* pAllUUIDs,            //[NumBuckets]
  const ValueType* __restrict__ pFallbackValue,  //[ValueSize]  
  BoolType* pIsFound,        //[NumKeys]
  ll_t numKeys, ll_t numBuckets
){
  constexpr int TPB = _TPB_;
  constexpr int KPT = _KPT_;
  constexpr int KeySize = _KEYSIZE_;
  constexpr int ValueSize = _VALUESIZE_;
  constexpr int KPB = TPB * KPT;

  int tid = threadIdx.x;
  ll_t kStart = blockIdx.x * KPB;

  ClosedHashmap<KeyType, ValueType, KeySize, ValueSize> hashmap(
    pPrime1, pPrime2,
    pAlpha1, pAlpha2,
    pBeta1,  pBeta2,
    pKeyPerm,
    pAllKeys,
    pAllValues,
    pAllUUIDs,
    numBuckets,
    -1,
    -3
  );

  // Load keys
  KeyType keys[KPT][KeySize];
  ValueType values[KPT][ValueSize];
  ValueType fallbackValue[ValueSize];
  #pragma unroll
  for (int i=0; i<KPT; i++){
    ll_t offset = kStart + i * TPB + tid;
    if (offset < numKeys){
      #pragma unroll
      for (int j=0; j<KeySize; j++){
        keys[i][j] = pKeys[offset * KeySize + j];
        // keys[i][j] = pKeys[offset * KeySize + hashmap.keyPerm[j]];
      }
    }
  }
  
  #pragma unroll
  for (int i=0; i<ValueSize; i++){
    fallbackValue[i] = pFallbackValue[i];
  }

  // get values
  bool isFound[KPT];
  // hashmap.get_batched<KPT>(keys, values, fallbackValue, isFound);
  #pragma unroll
  for (int i=0; i<KPT; i++){
    int offset = kStart + i * TPB + tid;
    if (offset < numKeys){
      isFound[i] = hashmap.get(keys[i], values[i], fallbackValue);

      pIsFound[offset] = (BoolType) isFound[i];
      if (isFound[i]){
        #pragma unroll
        for (int j=0; j<ValueSize; j++){
          pValues[offset * ValueSize + j] = values[i][j];
        }
      }
    }
  }
}

extern "C"
__global__ void closed_hashmap_set(
  const ll_t* __restrict__ pPrime1, //[KeySize]
  const ll_t* __restrict__ pPrime2, //[KeySize]
  const ll_t* __restrict__ pAlpha1, //[KeySize]
  const ll_t* __restrict__ pAlpha2, //[KeySize]
  const ll_t* __restrict__ pBeta1,  //[KeySize]
  const ll_t* __restrict__ pBeta2,  //[KeySize]
  const ll_t* __restrict__ pKeyPerm,             //[KeySize]
  const KeyType* __restrict__ pKeys,             //[NumKeys, KeySize]
  const ValueType* __restrict__ pValues,         //[NumKeys, ValueSize]
  KeyType* pAllKeys,          //[NumBuckets, KeySize]
  ValueType* pAllValues,      //[NumBuckets, ValueSize]
  ll_t* pAllUUIDs,            //[NumBuckets]
  BoolType* pIsStored,        //[NumKeys]
  ll_t numKeys, ll_t numBuckets
){
  constexpr int TPB = _TPB_;
  constexpr int KPT = _KPT_;
  constexpr int KeySize = _KEYSIZE_;
  constexpr int ValueSize = _VALUESIZE_;
  constexpr int KPB = TPB * KPT;

  int tid = threadIdx.x;
  ll_t kStart = blockIdx.x * KPB;

  ClosedHashmap<KeyType, ValueType, KeySize, ValueSize> hashmap(
    pPrime1, pPrime2,
    pAlpha1, pAlpha2,
    pBeta1,  pBeta2,
    pKeyPerm,
    pAllKeys,
    pAllValues,
    pAllUUIDs,
    numBuckets,
    -1,
    -3
  );

  // Load keys
  KeyType keys[KPT][KeySize];
  ValueType values[KPT][ValueSize];
  #pragma unroll
  for (int i=0; i<KPT; i++){
    ll_t offset = kStart + i * TPB + tid;
    if (offset < numKeys){
      #pragma unroll
      for (int j=0; j<KeySize; j++){
        keys[i][j] = pKeys[offset * KeySize + j];      
      }
      #pragma unroll
      for (int j=0; j<ValueSize; j++){
        values[i][j] = pValues[offset * ValueSize + j];
      }
    }
  }

  // get values
  bool isStored[KPT];
  // hashmap.set_batched<KPT>(keys, values, isStored);

  #pragma unroll
  for (int i=0; i<KPT; i++){
    int offset = kStart + i * TPB + tid;
    if (offset < numKeys){
      isStored[i] = hashmap.set(keys[i], values[i]);
      pIsStored[offset] = (BoolType) isStored[i];
    }
  }
}

extern "C"
__global__ void closed_hashmap_remove(
  const ll_t* __restrict__ pPrime1, //[KeySize]
  const ll_t* __restrict__ pPrime2, //[KeySize]
  const ll_t* __restrict__ pAlpha1, //[KeySize]
  const ll_t* __restrict__ pAlpha2, //[KeySize]
  const ll_t* __restrict__ pBeta1,  //[KeySize]
  const ll_t* __restrict__ pBeta2,  //[KeySize]
  const ll_t* __restrict__ pKeyPerm,             //[KeySize]
  const KeyType* __restrict__ pKeys,             //[NumKeys, KeySize]
  KeyType* pAllKeys,          //[NumBuckets, KeySize]
  ValueType* pAllValues,      //[NumBuckets, ValueSize]
  ll_t* pAllUUIDs,            //[NumBuckets]
  BoolType* pIsRemoved,        //[NumKeys]
  ll_t numKeys, ll_t numBuckets
){
  constexpr int TPB = _TPB_;
  constexpr int KPT = _KPT_;
  constexpr int KeySize = _KEYSIZE_;
  constexpr int ValueSize = _VALUESIZE_;
  constexpr int KPB = TPB * KPT;

  int tid = threadIdx.x;
  ll_t kStart = blockIdx.x * KPB;

  ClosedHashmap<KeyType, ValueType, KeySize, ValueSize> hashmap(
    pPrime1, pPrime2,
    pAlpha1, pAlpha2,
    pBeta1,  pBeta2,
    pKeyPerm,
    pAllKeys,
    pAllValues,
    pAllUUIDs,
    numBuckets,
    -1,
    -3
  );

  // Load keys
  KeyType keys[KPT][KeySize];
  #pragma unroll
  for (int i=0; i<KPT; i++){
    ll_t offset = kStart + i * TPB + tid;
    if (offset < numKeys){
      #pragma unroll
      for (int j=0; j<KeySize; j++){
        keys[i][j] = pKeys[offset * KeySize + j];
        // keys[i][j] = pKeys[offset * KeySize + hashmap.keyPerm[j]];
      }
    }
  }
  
  // remove
  bool isRemoved[KPT];
  #pragma unroll
  for (int i=0; i<KPT; i++){
    int offset = kStart + i * TPB + tid;
    if (offset < numKeys){
      isRemoved[i] = hashmap.remove(keys[i]);
      pIsRemoved[offset] = (BoolType) isRemoved[i];
    }
  }
}